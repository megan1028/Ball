#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <windows.h>  // for MS Windows
#include <GL/glut.h>  // GLUT, includes glu.h and gl.h
#include <Math.h>     // Needed for sin, cos
#include <thread>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <iostream>
#include <thread>

using namespace std;
#define PI 3.14159265f
#define PI 3.14159265f

#include <stdio.h>
#include <windows.h>  // for MS Windows
#include <GL/glut.h>  // GLUT, includes glu.h and gl.h
#include <Math.h>     // Needed for sin, cos
#include <thread>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <iostream>
#include <thread>
hipError_t initialWithCuda( unsigned int size);

float WINDOW_SIZE = 700;

int const NUM = 3;			//total number of circles (10 by default)
float radius = 0.1;			//circle radius
int waitTime = 33;			//milliseconds between steps
//coordinates
float x[NUM];
float y[NUM];
//velocity
float vx[NUM];
float vy[NUM];
//new velocity
float newvx[NUM];
float newvy[NUM];

//gravity
float g = 9.8;


float glowAlph[NUM];

//creates a gl triangle fan circle of indicated radius and segments
void flatCircle(float cx, float cy, float radius, int segments) {
	float phi, x1, y1;
	glBegin(GL_TRIANGLE_FAN);
	glVertex2f(cx, cy);					//center vertex
	for (int j = 0; j <= segments; j++) {	//for every segment,
		phi = 2 * PI * j / segments;	//calculate the new vertex
		x1 = radius * cos(phi) + cx;
		y1 = radius * sin(phi) + cy;
		glVertex2f(x1, y1);
	}
	glEnd();
} //end circle

__global__ void initialKernel(float *x, float* y, float *vx, float* vy, float rand1, float rand2, float radius)

{
	int i = threadIdx.x;
	//current position
	x[i] = (rand1 / 100.0) - (1.0 - radius); //  random number between
	y[i] = (rand1 / 100.0) - (1.0 - radius); //    -0.9 and 0.9 (to account for radius size)

	//velocity
	vx[i] = (rand2 / 10000.0) - 0.01; 	//	random velocities between
	vy[i] = (rand2 / 10000.0) - 0.01; 	//	  -0.02 and 0.02


}

//initializes all circle posiitons, colors, and velocities
void initCircles(void) {
	srand(time(NULL));							// seed the random number generator
	for (int i = 0; i < NUM; i++) {				// for each circle,
		//current position
		x[i] = ((rand() % (int)(200 - (radius * 200))) / 100.0) - (1.0 - radius); //  random number between
		y[i] = ((rand() % (int)(200 - (radius * 200))) / 100.0) - (1.0 - radius); //    -0.9 and 0.9 (to account for radius size)

		//velocity
		vx[i] = ((rand() % 200) / 10000.0) - 0.01; 	//	random velocities between
		vy[i] = ((rand() % 200) / 10000.0) - 0.01; 	//	  -0.02 and 0.02

		glowAlph[i] = 0.0;
	}
}



/* Callback handler for window re-paint event */
void display() {

	glClear(GL_COLOR_BUFFER_BIT);  // Clear the color buffer

	for (int i = 0; i < NUM; i++) {
		if (i % 3 == 0) {
			glColor3f(1.0, 0.0, 0.0);
			flatCircle(x[i] + x[i] / 15, y[i] + (y[i] - 1.0) / 20, radius, 30);
		}
		else if (i % 3 == 1) {
			glColor3f(0.0, 1.0, 0.0);
			flatCircle(x[i] + x[i] / 15, y[i] + (y[i] - 1.0) / 20, radius + 0.025, 30);
		}
		else {
			glColor3f(0.0, 0.0, 1.0);
			flatCircle(x[i] + x[i] / 15, y[i] + (y[i] - 1.0) / 20, radius + 0.04, 30);
		}

	}
	glFlush();
}


/* Call back when the windows is re-sized */
void reshape(int w, int h) {
	float aspectRatio = 1.0;

	//Compute the aspect ratio of the resized window
	aspectRatio = (float)h / (float)w;

	// Adjust the clipping box
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	if (h >= w)
		gluOrtho2D(-1.0, 1.0, -aspectRatio, aspectRatio);
	else
		gluOrtho2D(-1.0 / aspectRatio, 1.0 / aspectRatio, -1.0, 1.0);
	glMatrixMode(GL_MODELVIEW);

	//adjust the viewport
	glViewport(0, 0, w, h);
}


/* Called back when the timer expired */
void timer(int value) {
	//Actually move the circles
	for (int i = 0; i < NUM; i++) {
		x[i] += vx[i];
		y[i] += vy[i];
	}
	for (int i = 0; i < NUM; i++) {
		vy[i] = vy[i] - g * 0.0001 * value;
	}

	//resolve collisions
	for (int i = 0; i < NUM; i++) {	//for each ball,
		// Reverse direction when you reach edges
		if (x[i] > 1.0 - radius) {		//right edge
			x[i] = 1.0 - radius;				//to prevent balls from sticking
			vx[i] = -vx[i];					//change velocity

		}
		else if (x[i] < -1.0 + radius) {	//left edge
			x[i] = -1.0 + radius;				///to prevent balls from sticking
			vx[i] = -vx[i];					//change velocity

		}

		if (y[i] > 1.0 - radius) {		//top edge
			y[i] = 1.0 - radius;			//to prevent balls from sticking
			vy[i] = -vy[i];					//change velocity

		}
		else if (y[i] < -1.0 + radius) {	//bottom edge
			y[i] = -1.0 + radius;			//to prevent balls from sticking
			vy[i] = -vy[i];					//change velocity

		}
	}

	glutPostRedisplay();
	glutTimerFunc(waitTime, timer, 1);
}

/*__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}*/


int main(int argc, char **argv)
{
    /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };*/

    // Add vectors in parallel.
    hipError_t cudaStatus = initialWithCuda(NUM);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

   /* printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
*/
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
		
	//initCircles();		//initialize circle values

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_SINGLE | GLUT_RGBA | GLUT_ALPHA);

	glutInitWindowPosition(0, 0);					//window position
	glutInitWindowSize(WINDOW_SIZE, WINDOW_SIZE);	//window size
	glutCreateWindow("Bouncing balls");				//window name
	glClearColor(0.0, 0.0, 0.0, 0.0);				//background color
	glClear(GL_COLOR_BUFFER_BIT);

	//The four following statements set up the viewing rectangle
	glMatrixMode(GL_PROJECTION);					// use proj. matrix
	glLoadIdentity();								// load identity matrix
	gluOrtho2D(-1.0, 1.0, -1.0, 1.0);				// set orthogr. proj.
	glMatrixMode(GL_MODELVIEW);						// back to modelview m.

	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	glutDisplayFunc(display);
	
	glutTimerFunc(waitTime, timer, 1);
	glutReshapeFunc(reshape);

	glutMainLoop();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t initialWithCuda( float *a, float *b, float *c, float *d, unsigned int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
       // goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
       //goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
       // goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    } 

    // Copy input vectors from host memory to GPU buffers.
  /*  cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    } */

    // Launch a kernel on the GPU with one thread for each element.
	float rand1 = rand() % (int)(200 - (radius * 200));
	float rand2 = rand() % 200;
    initialKernel<<<1, size>>>(rand1, rand2, radius);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
       // goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
      ///  goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
  /*  cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/

//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
    
    return cudaStatus;
}
